#include "hip/hip_runtime.h"
/*   Linear Intepolation Demo 
 *    
 *    Copyright (C) 2012-2013 Orange Owl Solutions.  
 *
 *    This file is part of  Linear Intepolation Demo
 *    Linear Intepolation Demo is free software: you can redistribute it and/or modify
 *    it under the terms of the Lesser GNU General Public License as published by
 *    the Free Software Foundation, either version 3 of the License, or
 *    (at your option) any later version.
 *
 *    Bluebird Library is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *    Lesser GNU General Public License for more details.
 *
 *    You should have received a copy of the GNU General Public License
 *    along with Linear Intepolation Demo.  If not, see <http://www.gnu.org/licenses/>.
 *
 *
 *    For any request, question or bug reporting please visit http://www.orangeowlsolutions.com/
 *    or send an e-mail to: info@orangeowlsolutions.com
 *
 *
 */

// includes, system
#include <cstdlib> 
#include <conio.h>
#include <math.h>
#include <fstream>
#include <iostream> 
#include <iomanip>

// includes, cuda 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

texture<float, 1, hipReadModeElementType> data_d_texture_filtering;
texture<float, 1> data_d_texture;

#define BLOCK_SIZE 256

/******************/
/* ERROR CHECKING */
/******************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) { getch(); exit(code); }
  }
}
/************/
/* LINSPACE */
/************/
// --- Generates N equally spaced, increasing points between a and b and stores them in x 
void linspace(float* x, float a, float b, int N) {
  float delta_x=(b-a)/(float)N;
  x[0]=a;
  for(int k=1;k<N;k++) x[k]=x[k-1]+delta_x;
}

/*************/
/* RANDSPACE */
/*************/
// --- Generates N randomly spaced, increasing points between a and b and stores them in x 
void randspace(float* x, float a, float b, int N) {
  float delta_x=(b-a)/(float)N;
  x[0]=a;
  for(int k=1;k<N;k++) x[k]=x[k-1]+delta_x+(((float)rand()/(float)RAND_MAX-0.5)*(1./(float)N));
}

/******************/
/* DATA GENERATOR */
/******************/
// --- Generates N complex random data points, with real and imaginary parts ranging in (0.f,1.f)
void Data_Generator(float* data, int N) {
  for(int k=0;k<N;k++) {
    data[k]=(float)rand()/(float)RAND_MAX;
  }
}

/*************************************/
/* LINEAR INTERPOLATION KERNEL - CPU */
/*************************************/
float linear_kernel_CPU(float in)
{
  float d_y;
  return 1.-abs(in);
}

/***************************************/
/* LINEAR INTERPOLATION FUNCTION - CPU */
/***************************************/
void linear_interpolation_function_CPU(float* result_GPU, float* data, float* x_in, float* x_out, int M, int N){

  float a;
  for(int j=0; j<N; j++){
    int k = floor(x_out[j]+M/2);
    a = x_out[j]+M/2-floor(x_out[j]+M/2);
    result_GPU[j] = a * data[k+1] + (-data[k] * a + data[k]);
  }	

}

/*************************************/
/* LINEAR INTERPOLATION KERNEL - GPU */
/*************************************/
__device__ float linear_kernel_GPU(float in)
{
  float d_y;
  return 1.-abs(in);
}

/**************************************************************/
/* LINEAR INTERPOLATION KERNEL FUNCTION - GPU - GLOBAL MEMORY */
/**************************************************************/
__global__ void linear_interpolation_kernel_function_GPU(float* __restrict__ result_d, const float* __restrict__ data_d, const float* __restrict__ x_out_d, const int M, const int N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;

  if(j<N)
  {
    float reg_x_out = x_out_d[j]+M/2;
    int k = __float2int_rz(reg_x_out); 
    float a = reg_x_out - truncf(reg_x_out);
    float dk = data_d[k];
    float dkp1 = data_d[k+1];
    result_d[j] = a * dkp1 + (-dk * a + dk);
  } 
}

/***************************************************************/
/* LINEAR INTERPOLATION KERNEL FUNCTION - GPU - TEXTURE MEMORY */
/***************************************************************/
__global__ void linear_interpolation_kernel_function_GPU_texture(float* __restrict__ result_d, const float* __restrict__ x_out_d, const int M, const int N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;

  if(j<N)
  {
    float reg_x_out = x_out_d[j]+M/2;
    int k = __float2int_rz(reg_x_out); 
    float a = reg_x_out - truncf(reg_x_out);
    float dk = tex1Dfetch(data_d_texture,k);
    float dkp1 = tex1Dfetch(data_d_texture,k+1);
    result_d[j] = a * dkp1 + (-dk * a + dk);
  } 
}

__global__ void linear_interpolation_kernel_function_GPU_texture_filtering(float* __restrict__ result_d, const float* __restrict__ x_out_d, const int M, const int N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x; 
  if(j<N) result_d[j] = tex1D(data_d_texture_filtering,float(x_out_d[j]+M/2+0.5));
}

/***************************************/
/* LINEAR INTERPOLATION FUNCTION - GPU */
/***************************************/
void linear_interpolation_function_GPU(float* result_d, float* data_d, float* x_in_d, float* x_out_d, int M, int N){

  dim3 dimBlock(BLOCK_SIZE,1); dim3 dimGrid(N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1),1);
  linear_interpolation_kernel_function_GPU<<<dimGrid,dimBlock>>>(result_d, data_d, x_out_d, M, N);
}

/*************************************************/
/* LINEAR INTERPOLATION FUNCTION - GPU - TEXTURE */
/*************************************************/
void linear_interpolation_function_GPU_texture(float* result_d, float* data_d, float* x_in_d, float* x_out_d, int M, int N){

  hipBindTexture(NULL, data_d_texture, data_d, M*sizeof(float));

  dim3 dimBlock(BLOCK_SIZE,1); dim3 dimGrid(N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1),1);
  linear_interpolation_kernel_function_GPU_texture<<<dimGrid,dimBlock>>>(result_d, x_out_d, M, N);
}

void linear_interpolation_function_GPU_texture_filtering(float* result_d, hipArray* data_d_cudaArray, float* x_in_d, float* x_out_d, int M, int N){

  gpuErrchk(hipBindTextureToArray(data_d_texture_filtering, data_d_cudaArray));
  data_d_texture_filtering.normalized = false;
  data_d_texture_filtering.filterMode = hipFilterModeLinear;

  dim3 dimBlock(BLOCK_SIZE,1); dim3 dimGrid(N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1),1);
  linear_interpolation_kernel_function_GPU_texture_filtering<<<dimGrid,dimBlock>>>(result_d, x_out_d, M, N);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
}

/********/
/* MAIN */
/********/
int main()
{
  // --- Number of input points
  // int M=1024*256;
  int M=1024*256;

  // --- Number of output points
  // int N=1024*256;
  int N=1024*256;

  // --- Number of computations for time measurement
  int Nit = 100;

  // --- Input sampling
  float* x_in=(float*)malloc(M*sizeof(float));
  float* x_in_d;		gpuErrchk(hipMalloc((void**)&x_in_d,sizeof(float)*M));
  linspace(x_in,-M/2.,M/2.,M);
  gpuErrchk(hipMemcpy(x_in_d,x_in,sizeof(float)*M,hipMemcpyHostToDevice));

  // --- Input data
  float *data;		data=(float*)malloc((M+1)*sizeof(float));
  float* data_d;		gpuErrchk(hipMalloc((void**)&data_d,sizeof(float)*(M+1)));
  hipMemset(data_d,0,sizeof(float)*(M+1));
  Data_Generator(data,M);
  data[M]=0.;
  gpuErrchk(hipMemcpy(data_d,data,sizeof(float)*M,hipMemcpyHostToDevice));

  // --- Output sampling
  float* x_out;		x_out=(float*)malloc(N*sizeof(float));
  float* x_out_d;		gpuErrchk(hipMalloc((void**)&x_out_d,sizeof(float)*N));
  randspace(x_out,-M/2.,M/2.,N);
  gpuErrchk(hipMemcpy(x_out_d,x_out,sizeof(float)*N,hipMemcpyHostToDevice));

  //hipArray* data_d_cudaArray = NULL; gpuErrchk(hipMallocArray (&data_d_cudaArray, &data_d_texture_filtering.channelDesc, M, 1));
  //gpuErrchk(hipMemcpyToArray(data_d_cudaArray, 0, 0, data, sizeof(float)*M, hipMemcpyHostToDevice));

  // --- Result allocation
  float *result_GPU;							result_GPU=(float*)malloc(N*sizeof(float));
  float *result_texture;						result_texture=(float*)malloc(N*sizeof(float));
  float *result_texture_filtering;			result_texture_filtering=(float*)malloc(N*sizeof(float));
  float *result_CPU;							result_CPU=(float*)malloc(N*sizeof(float));
  float *result_d;							gpuErrchk(hipMalloc((void**)&result_d,sizeof(float)*N));
  float *result_d_texture;					gpuErrchk(hipMalloc((void**)&result_d_texture,sizeof(float)*N));
  float *result_d_texture_filtering;			gpuErrchk(hipMalloc((void**)&result_d_texture_filtering,sizeof(float)*N));

  // --- Reference interpolation result as evaluated on the CPU
  linear_interpolation_function_CPU(result_CPU, data, x_in, x_out, M, N);

  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  for (int k=0; k<Nit; k++) linear_interpolation_function_GPU(result_d, data_d, x_in_d, x_out_d, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Global memory [ms]: " << setprecision (10) << time/Nit << endl;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  //for (int k=0; k<Nit; k++) linear_interpolation_function_GPU_texture_filtering(result_d_texture_filtering, data_d_cudaArray, x_in_d, x_out_d, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Texture filtering [ms]: " << setprecision (10) << time/Nit << endl;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  for (int k=0; k<Nit; k++) linear_interpolation_function_GPU_texture(result_d_texture, data_d, x_in_d, x_out_d, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Texture [ms]: " << setprecision (10) << time/Nit << endl;

  gpuErrchk(hipMemcpy(result_GPU,result_d,sizeof(float)*N,hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(result_texture_filtering,result_d_texture_filtering,sizeof(float)*N,hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(result_texture,result_d_texture,sizeof(float)*N,hipMemcpyDeviceToHost));

  float diff_norm=0.f, norm=0.f;
  for(int j=0; j<N; j++) {
    diff_norm = diff_norm + (result_CPU[j]-result_GPU[j])*(result_CPU[j]-result_GPU[j]);
    norm      = norm      + result_CPU[j]*result_CPU[j];
    //printf("%f %f\n",result_CPU[j],result_GPU[j]);
  }
  printf("Error GPU [percentage] = %f\n",100.*sqrt(diff_norm/norm));

  float diff_norm_texture_filtering=0.f;
  for(int j=0; j<N; j++) {
    diff_norm_texture_filtering = diff_norm_texture_filtering + (result_CPU[j]-result_texture_filtering[j])*(result_CPU[j]-result_texture_filtering[j]);
  }
  printf("Error texture filtering [percentage] = %f\n",100.*sqrt(diff_norm_texture_filtering/norm));

  float diff_norm_texture=0.f;
  for(int j=0; j<N; j++) {
    diff_norm_texture = diff_norm_texture + (result_CPU[j]-result_texture[j])*(result_CPU[j]-result_texture[j]);
  }
  printf("Error texture [percentage] = %f\n",100.*sqrt(diff_norm_texture/norm));

  hipDeviceReset();

  ofstream outfile;
  outfile.open("x_in.dat", ios::out | ios::binary);							for(int i=0; i<M; i++){ outfile.write( (char*)&x_in[i], sizeof(float)); } outfile.close();
  outfile.open("data.dat", ios::out | ios::binary);							for(int i=0; i<M; i++){ outfile.write( (char*)&data[i], sizeof(float)); } outfile.close();
  outfile.open("x_out.dat", ios::out | ios::binary);							for(int i=0; i<N; i++){ outfile.write( (char*)&x_out[i], sizeof(float)); } outfile.close();
  outfile.open("result_texture_filtering.dat", ios::out | ios::binary);		for(int i=0; i<N; i++){ outfile.write( (char*)&result_texture_filtering[i], sizeof(float)); } outfile.close();
  outfile.open("result_texture.dat", ios::out | ios::binary);					for(int i=0; i<N; i++){ outfile.write( (char*)&result_texture[i], sizeof(float)); } outfile.close();
  outfile.open("result_GPU.dat", ios::out | ios::binary);						for(int i=0; i<N; i++){ outfile.write( (char*)&result_GPU[i], sizeof(float)); } outfile.close();
  outfile.open("result_CPU.dat", ios::out | ios::binary);						for(int i=0; i<N; i++){ outfile.write( (char*)&result_CPU[i], sizeof(float)); } outfile.close();

  getch();

  return 0;
}


