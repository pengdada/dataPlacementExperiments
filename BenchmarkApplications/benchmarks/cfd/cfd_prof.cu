#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#include "../include/common.h"
#define K 1
using namespace std;

//#define cfd_SUPER_BLOCKS_PER_SM 5
//const int BLOCK_SIZE = 256;
const int cfd_nBlksPerCluster = 16; 
const int cfd_nAtom = BLOCK_SIZE * MSIZE;
const int cfd_maxNeighbors = 8;


inline int * cfd_myBuildNeighborList_blkSchedule(const int nAtom, 
        int* neighborList, int blockSz)
{
  //create non-uniform data sharing
  //but avoid that tasks sharing the same data are neighbor tasks by randomization
  vector<int> atomInds(nAtom);
  vector<int> blkInds((nAtom+blockSz-1)/blockSz);
  for(int i=0; i<blkInds.size(); ++i)
    blkInds[i] = i;
  random_shuffle(blkInds.begin(), blkInds.end());

  int *blkOrder = (int*)malloc(blkInds.size()*sizeof(int));
  for(int i=0; i<blkInds.size(); ++i)
    blkOrder[i] = blkInds[i];

  int j=0; 
  for(vector<int>::iterator it=blkInds.begin(); it!=blkInds.end(); ++it)
  {
    int blkInd = *it;
    for(int i=0; i<blockSz; ++i)
      atomInds[j++] = blkInd*blockSz + i;
  }
  int superBlockSz = blockSz * cfd_nBlksPerCluster;
  // Build Neighbor List
  for (int i = 0; i < nAtom; i++)
  {
    int start = i - i%superBlockSz; //difference is here
    //int end = i + (superBlockSz - i%superBlockSz)-1;

    int nNeighbors = 0;
    do {
      int j = start + rand() % superBlockSz;
      if (i == j || j>=nAtom) continue; // An atom cannot be its own neighbor
      neighborList[nNeighbors*nAtom + atomInds[i]] = atomInds[j];
      nNeighbors ++; 
    } while(nNeighbors<cfd_maxNeighbors);

  }
  return blkOrder;
}

#define GAMMA 1.4f
#define VAR_DENSITY 0
#define VAR_MOMENTUM 1
#define NDIM 3
#define VAR_DENSITY_ENERGY (VAR_MOMENTUM+NDIM)
#define NVAR (VAR_DENSITY_ENERGY+1)

__host__ __device__ inline void compute_velocity(float& density, float3& momentum, float3& velocity)
{
	velocity.x = momentum.x / density;
	velocity.y = momentum.y / density;
	velocity.z = momentum.z / density;
}

__host__ __device__ inline float compute_speed_sqd(float3& velocity)
{
	return velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z;
}

__host__ __device__ inline float compute_pressure(float& density, float& density_energy, float& speed_sqd)
{
	return (float(GAMMA)-float(1.0f))*(density_energy - float(0.5f)*density*speed_sqd);
}

__host__ __device__ inline float compute_speed_of_sound(float& density, float& pressure)
{
	return sqrtf(float(GAMMA)*pressure/density);
}
__host__ __device__ __host__ inline void compute_flux_contribution(float& density, float3& momentum, float& density_energy, float& pressure, float3& velocity, float3& fc_momentum_x, float3& fc_momentum_y, float3& fc_momentum_z, float3& fc_density_energy)
{
	fc_momentum_x.x = velocity.x*momentum.x + pressure;
	fc_momentum_x.y = velocity.x*momentum.y;
	fc_momentum_x.z = velocity.x*momentum.z;
	
	
	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y*momentum.y + pressure;
	fc_momentum_y.z = velocity.y*momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z*momentum.z + pressure;

	float de_p = density_energy+pressure;
	fc_density_energy.x = velocity.x*de_p;
	fc_density_energy.y = velocity.y*de_p;
	fc_density_energy.z = velocity.z*de_p;
}

void check_cfd(int nelr, int* elements_surrounding_elements, float*
    normals, float* density, float* mx, float* my, float* mz, float* density_energy, float* fluxes)
    {
      const float smoothing_coefficient = float(0.2f);
	//const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	for(int i=0;i<MSIZE*BLOCK_SIZE;i++){
	int j, nb;
	float3 normal; float normal_len;
	float factor;
	
	//float density_i = variables[i + VAR_DENSITY*nelr];  
	float density_i = density[i];
	float3 momentum_i;
	//momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	//momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	//momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];
  momentum_i.x = mx[i];
  momentum_i.y = my[i];
  momentum_i.z = mz[i];

	//float density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];
	float density_energy_i = density_energy[i];

	float3 velocity_i;             				compute_velocity(density_i, momentum_i, velocity_i);
	float speed_sqd_i                          = compute_speed_sqd(velocity_i);
	float speed_i                              = sqrtf(speed_sqd_i);
	float pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
	float speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
	float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
	float3 flux_contribution_i_density_energy;	
	compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);
	
	//float flux_i_density = float(0.0f);
	float flux_i_density = 0.0;
	float3 flux_i_momentum;
	flux_i_momentum.x = float(0.0f);
	flux_i_momentum.y = float(0.0f);
	flux_i_momentum.z = float(0.0f);
	float flux_i_density_energy = float(0.0f);
		
	float3 velocity_nb;
	float density_nb, density_energy_nb;
	float3 momentum_nb;
	float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
	float3 flux_contribution_nb_density_energy;	
	float speed_sqd_nb, speed_of_sound_nb, pressure_nb;
	
	#pragma unroll
	for(j = 0; j < cfd_maxNeighbors; j++)
	{
		nb = elements_surrounding_elements[i + j*nelr];
    //optimal layout already 
    // |X for neighbor 0, X for neighbor 1, ... | Y for neighbor 0, Y for neighbor 1, ...
    // |Z for neighbor 0, Z for neighbor 1, ... |
		normal.x = normals[i + (j + 0*cfd_maxNeighbors)*nelr]; 
		normal.y = normals[i + (j + 1*cfd_maxNeighbors)*nelr];
		normal.z = normals[i + (j + 2*cfd_maxNeighbors)*nelr];
		normal_len = sqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
		
		if(nb >= 0) 	// a legitimate neighbor
		{
			//density_nb = variables[nb + VAR_DENSITY*nelr];
			//momentum_nb.x = variables[nb + (VAR_MOMENTUM+0)*nelr];
			//momentum_nb.y = variables[nb + (VAR_MOMENTUM+1)*nelr];
			//momentum_nb.z = variables[nb + (VAR_MOMENTUM+2)*nelr];
			density_nb = density[nb];
			momentum_nb.x = mx[nb];
			momentum_nb.y = my[nb];
			momentum_nb.z = mz[nb];
			//density_energy_nb = variables[nb + VAR_DENSITY_ENERGY*nelr];
			density_energy_nb = density_energy[nb];
												compute_velocity(density_nb, momentum_nb, velocity_nb);
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
			                                    compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
			
			// artificial viscosity
			//factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
      factor = 1.3;
			flux_i_density += factor*(density_i-density_nb);
			flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
			flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
			flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
			flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(momentum_nb.x+momentum_i.x);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(momentum_nb.y+momentum_i.y);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
			
			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(momentum_nb.z+momentum_i.z);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
		}
	}

		/*if(((pow((fluxes[i + VAR_DENSITY*nelr] - flux_i_density),2)/flux_i_density)>0.001)||\
          ((pow((fluxes[i + (VAR_MOMENTUM+0)*nelr] - flux_i_momentum.x),2)/flux_i_momentum.x)>0.001)||\
          ((pow((fluxes[i + (VAR_MOMENTUM+1)*nelr] - flux_i_momentum.y),2)/flux_i_momentum.y)>0.001)||\
          ((pow((fluxes[i + (VAR_MOMENTUM+2)*nelr] - flux_i_momentum.z),2)/flux_i_momentum.z)>0.001)||\
          ((pow((fluxes[i + VAR_DENSITY_ENERGY*nelr]- flux_i_density_energy),2)/flux_i_density_energy)>0.001))*/
          
     if(((abs((fluxes[i + VAR_DENSITY*nelr] - flux_i_density)/flux_i_density)>0.01)&&(abs((fluxes[i + VAR_DENSITY*nelr] - flux_i_density))>0.01))||\
                  ((abs((fluxes[i + (VAR_MOMENTUM+0)*nelr] - flux_i_momentum.x)/flux_i_momentum.x)>0.01)&&(abs((fluxes[i + (VAR_MOMENTUM+0)*nelr] - flux_i_momentum.x))>0.01))||\
          ((abs((fluxes[i + (VAR_MOMENTUM+1)*nelr] - flux_i_momentum.y)/flux_i_momentum.y)>0.01)&&(abs((fluxes[i + (VAR_MOMENTUM+1)*nelr] - flux_i_momentum.y))>0.01))||\
          ((abs((fluxes[i + (VAR_MOMENTUM+2)*nelr] - flux_i_momentum.z)/flux_i_momentum.z)>0.01)&&(abs((fluxes[i + (VAR_MOMENTUM+2)*nelr] - flux_i_momentum.z))>0.01))||\
          ((abs((fluxes[i + VAR_DENSITY_ENERGY*nelr]- flux_i_density_energy)/flux_i_density_energy)>0.01)&&(abs((fluxes[i + VAR_DENSITY_ENERGY*nelr]- flux_i_density_energy))>0.01)))
              {printf("failed!%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",i,fluxes[i + VAR_DENSITY*nelr],flux_i_density,\
                                                                 fluxes[i + (VAR_MOMENTUM+0)*nelr],flux_i_momentum.x,\
                                                                 fluxes[i + (VAR_MOMENTUM+1)*nelr] , flux_i_momentum.y,\
                                                                 fluxes[i + (VAR_MOMENTUM+2)*nelr],flux_i_momentum.z,\
                                                                 fluxes[i + VAR_DENSITY_ENERGY*nelr],flux_i_density_energy);
	       return;} 
	 }
	 printf("GOOD! passed!\n");
	 return;
  }
__global__ void cfd_kernel(int nelr, int* elements_surrounding_elements, float*
    normals, float* density, float* mx, float* my, float* __restrict__  mz, float* density_energy, float* fluxes,int *d_flag)
{
	const float smoothing_coefficient = float(0.2f);
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	
	int j, nb;
	float3 normal; float normal_len;
	float factor;
	
	//float density_i = variables[i + VAR_DENSITY*nelr];  
	float density_i = density[i];
if(blockIdx.x == 0){
 printf("0 0 0 0 %d %d\n",threadIdx.x,i);}
	float3 momentum_i;
	//momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	//momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	//momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];
  momentum_i.x = mx[i];
  momentum_i.y = my[i];
  momentum_i.z = mz[i];
if(blockIdx.x == 0){
 printf("1 0 0 0 %d %d\n",threadIdx.x,i);
 printf("2 0 0 0 %d %d\n",threadIdx.x,i);
printf("3 0 0 0 %d %d\n",threadIdx.x,i);}
	//float density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];
	float density_energy_i = density_energy[i];
if(blockIdx.x == 0){
 printf("4 0 0 0 %d %d\n",threadIdx.x,i);}
	float3 velocity_i;             				compute_velocity(density_i, momentum_i, velocity_i);
	float speed_sqd_i                          = compute_speed_sqd(velocity_i);
	float speed_i                              = sqrtf(speed_sqd_i);
	float pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
	float speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
	float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
	float3 flux_contribution_i_density_energy;	
	compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);
	
	//float flux_i_density = float(0.0f);
	float flux_i_density = 0.0;
	float3 flux_i_momentum;
	flux_i_momentum.x = float(0.0f);
	flux_i_momentum.y = float(0.0f);
	flux_i_momentum.z = float(0.0f);
	float flux_i_density_energy = float(0.0f);
		
	float3 velocity_nb;
	float density_nb, density_energy_nb;
	float3 momentum_nb;
	float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
	float3 flux_contribution_nb_density_energy;	
	float speed_sqd_nb, speed_of_sound_nb, pressure_nb;
	
	#pragma unroll
	for(j = 0; j < cfd_maxNeighbors; j++)
	{
		nb = elements_surrounding_elements[i + j*nelr];
         if(blockIdx.x == 0){
 printf("5 0 0 %d %d %d\n",j,threadIdx.x,i+j*nelr);}
    //optimal layout already 
    // |X for neighbor 0, X for neighbor 1, ... | Y for neighbor 0, Y for neighbor 1, ...
    // |Z for neighbor 0, Z for neighbor 1, ... |
		normal.x = normals[i + (j + 0*cfd_maxNeighbors)*nelr]; 
		normal.y = normals[i + (j + 1*cfd_maxNeighbors)*nelr];
		normal.z = normals[i + (j + 2*cfd_maxNeighbors)*nelr];
 if(blockIdx.x == 0){
 printf("6 0 0 %d %d %d\n",j,threadIdx.x,i+j*nelr);
 printf("6 0 1 %d %d %d\n",j,threadIdx.x,i+(j+cfd_maxNeighbors)*nelr);
printf("6 0 2 %d %d %d\n",j,threadIdx.x,i+(j+2*cfd_maxNeighbors)*nelr);}
		normal_len = sqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
		
		if(nb >= 0) 	// a legitimate neighbor
		{
			//density_nb = variables[nb + VAR_DENSITY*nelr];
			//momentum_nb.x = variables[nb + (VAR_MOMENTUM+0)*nelr];
			//momentum_nb.y = variables[nb + (VAR_MOMENTUM+1)*nelr];
			//momentum_nb.z = variables[nb + (VAR_MOMENTUM+2)*nelr];
if(blockIdx.x == 0){
 printf("0 0 1 %d %d %d\n",j,threadIdx.x,nb);
printf("1 0 1 %d %d %d\n",j,threadIdx.x,nb);
printf("2 0 1 %d %d %d\n",j,threadIdx.x,nb);
printf("3 0 1 %d %d %d\n",j,threadIdx.x,nb);
printf("4 0 1 %d %d %d\n",j,threadIdx.x,nb);
}
			density_nb = density[nb];
			momentum_nb.x = mx[nb];
			momentum_nb.y = my[nb];
			momentum_nb.z = mz[nb];
			//density_energy_nb = variables[nb + VAR_DENSITY_ENERGY*nelr];
			density_energy_nb = density_energy[nb];
												compute_velocity(density_nb, momentum_nb, velocity_nb);
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
			                                    compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
			
			// artificial viscosity
			//factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
      factor = 1.3;
			flux_i_density += factor*(density_i-density_nb);
			flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
			flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
			flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
			flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(momentum_nb.x+momentum_i.x);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(momentum_nb.y+momentum_i.y);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
			
			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(momentum_nb.z+momentum_i.z);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
		}
	}

	fluxes[i + VAR_DENSITY*nelr] = flux_i_density;
	fluxes[i + (VAR_MOMENTUM+0)*nelr] = flux_i_momentum.x;
	fluxes[i + (VAR_MOMENTUM+1)*nelr] = flux_i_momentum.y;
	fluxes[i + (VAR_MOMENTUM+2)*nelr] = flux_i_momentum.z;
	fluxes[i + VAR_DENSITY_ENERGY*nelr] = flux_i_density_energy;
//if (threadIdx.x==0) atomicAdd(d_flag,1);
if(blockIdx.x == 0){
 printf("7 1 0 0 %d %d\n",threadIdx.x,i + VAR_DENSITY*nelr);
 printf("7 1 1 0 %d %d\n",threadIdx.x,i + (VAR_MOMENTUM+0)*nelr);
 printf("7 1 2 0 %d %d\n",threadIdx.x,i + (VAR_MOMENTUM+1)*nelr);
 printf("7 1 3 0 %d %d\n",threadIdx.x,i + (VAR_MOMENTUM+2)*nelr);
 printf("7 1 4 0 %d %d\n",threadIdx.x,i +VAR_DENSITY_ENERGY*nelr);
}

}

int main(int argc, char **argv) {
     size_t limit = 1024*1024*1024;
   hipDeviceSetLimit(hipLimitPrintfFifoSize,limit);  
srand(2013);
  // Allocate problem data on host
  //posVecType*   position;
  //forceVecType* force;
  float *density;
  float *mx;
  float *my;
  float *mz;
  float *density_energy;
  float *normals;
  float *fluxes;
  int* cfd_neighborList;

  hipHostMalloc((void**)&density, cfd_nAtom*sizeof(float));
  hipHostMalloc((void**)&mx, cfd_nAtom*sizeof(float));
  hipHostMalloc((void**)&my, cfd_nAtom*sizeof(float));
  hipHostMalloc((void**)&mz, cfd_nAtom*sizeof(float));
  hipHostMalloc((void**)&density_energy, cfd_nAtom*sizeof(float));
  hipHostMalloc((void**)&normals, cfd_nAtom*NDIM*cfd_maxNeighbors*sizeof(float));
  hipHostMalloc((void**)&fluxes, cfd_nAtom*NVAR*sizeof(float));
  hipHostMalloc((void**)&cfd_neighborList, cfd_nAtom*cfd_maxNeighbors*sizeof(int));

  // Allocate device memory for position and force
  //forceVecType* d_force;
  //posVecType*   d_position;
  float *d_density;
  float *d_mx;
  float *d_my;
  float *d_mz;
  float *d_density_energy;
  float *d_normals;
  float *d_fluxes;
  hipMalloc((void**)&d_density, cfd_nAtom*sizeof(float));
  hipMalloc((void**)&d_mx, cfd_nAtom*sizeof(float));
  hipMalloc((void**)&d_my, cfd_nAtom*sizeof(float));
  hipMalloc((void**)&d_mz, cfd_nAtom*sizeof(float));
  hipMalloc((void**)&d_density_energy, cfd_nAtom*sizeof(float));
  hipMalloc((void**)&d_normals, cfd_nAtom*NDIM*cfd_maxNeighbors*sizeof(float));
  hipMalloc((void**)&d_fluxes, cfd_nAtom*NVAR*sizeof(float));
  hipMemset(d_fluxes, 0, cfd_nAtom*NVAR*sizeof(float));
  //hipMemset(d_force, 0, cfd_nAtom*sizeof(forceVecType));

  // Allocate device memory for neighbor list
  int* d_cfd_neighborList;
  hipMalloc((void**)&d_cfd_neighborList, cfd_nAtom*cfd_maxNeighbors*sizeof(int));
printf("%d %d %d %d %d %d %d %d\n",cfd_nAtom*sizeof(float),cfd_nAtom*sizeof(float),cfd_nAtom*sizeof(float),cfd_nAtom*sizeof(float),cfd_nAtom*sizeof(float),cfd_nAtom*NDIM*cfd_maxNeighbors*sizeof(float),cfd_nAtom*NVAR*sizeof(float),cfd_nAtom*cfd_maxNeighbors*sizeof(int));
  //cout << "Initializing test problem (this can take several "
   //       "minutes for large problems)\n";

  // Initialize positions -- random distribution in cubic domain
  // domainEdge constant specifies edge length
  for (int i = 0; i < cfd_nAtom; i++)
  {
      density[i] = (float)(drand48());
      density_energy[i] = (float)(drand48() );
      mx[i] = (float)(drand48() );
      my[i] = (float)(drand48() );
      mz[i] = (float)(drand48() );
    /*
      density[i] = 1.1+i*0.01;

      density_energy[i] = 1.1+i*0.01;
      mx[i] = 1.1+i*0.01;
      my[i] = 1.1+i*0.01;
      mz[i] = 1.1+i*0.01;
      */
  }
  for(int i=0; i<cfd_nAtom*NDIM*cfd_maxNeighbors; ++i)
    normals[i] = (float)(drand48());

  cfd_myBuildNeighborList_blkSchedule(cfd_nAtom, cfd_neighborList, BLOCK_SIZE);

  hipMemcpy(d_cfd_neighborList, cfd_neighborList, cfd_maxNeighbors*cfd_nAtom*sizeof(int), hipMemcpyHostToDevice);
  // Copy data to GPU
  hipMemcpy(d_density, density, cfd_nAtom*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mx, mx, cfd_nAtom*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_my, my, cfd_nAtom*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_mz, mz, cfd_nAtom*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_density_energy, density_energy, cfd_nAtom*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_normals, normals, cfd_nAtom*NDIM*cfd_maxNeighbors*sizeof(float), hipMemcpyHostToDevice);
hipSetDeviceFlags(hipDeviceMapHost);
  int *flag_cfd,*d_flag_cfd;
  hipHostAlloc((void**)&flag_cfd,sizeof( int),hipHostMallocMapped);
  hipHostGetDevicePointer((void**)&d_flag_cfd,(void*)flag_cfd,0);

  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);

  int cfd_gridSize  = (cfd_nAtom-1+BLOCK_SIZE) / BLOCK_SIZE;

  cfd_kernel<<<cfd_gridSize, BLOCK_SIZE>>>(cfd_nAtom, d_cfd_neighborList, d_normals, d_density, d_mx, d_my, d_mz, d_density_energy,
      d_fluxes,d_flag_cfd);


  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time/ITERATIONS << endl;

  hipMemcpy(fluxes, d_fluxes, cfd_nAtom*NVAR*sizeof(float), hipMemcpyDeviceToHost);
  check_cfd(cfd_nAtom,cfd_neighborList,normals,density,mx,my,mz,density_energy,fluxes);
  
  //TODO:verified on small inputs
  /*
  ifstream fluxesF("../org/fluxes.txt");
  for(int i=0; i<cfd_nAtom*NVAR; ++i) {
    float f;
    fluxesF >> f;
    if(abs(f - fluxes[i]) > 0.001) {
      fprintf(stderr, "Test failed! i = %d\n", i);
      return 1;
    }
  }*/
 // printf("Test passed!\n");
 // fluxesF.close();


  return 0;
}

