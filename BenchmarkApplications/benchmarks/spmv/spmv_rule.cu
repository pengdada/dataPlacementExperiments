#include "hip/hip_runtime.h"
#include "config.h"

texture<float,1,hipReadModeElementType> tex_val;
texture<int,1,hipReadModeElementType> tex_col;
texture<float,1,hipReadModeElementType> tex_vec;
texture<int,1,hipReadModeElementType> tex_row;

__global__ void 
spmv_kernel(const float* val,
                       const int    * cols,
                       const int    * rowDelimiters,
                       const float  * vec,
                       const int dim, float * out)
{
  // Thread ID in block
  int t = threadIdx.x; 
  // Thread ID within warp
  int id = t & (WARP_SIZE-1);
  int warpsPerBlock = blockDim.x / WARP_SIZE;
  // One row per warp
  int myRow = (blockIdx.x * warpsPerBlock) + (t / WARP_SIZE);
  __shared__ volatile float partialSums[BLOCK_SIZE];

  if (myRow < dim) 
  {
    int warpStart =tex1Dfetch(tex_row,myRow);
    int warpEnd = tex1Dfetch(tex_row,myRow+1);
    float mySum = 0;
    for (int j = warpStart + id; j < warpEnd; j += WARP_SIZE)
    {
      int col = tex1Dfetch(tex_col,j); 
      mySum += tex1Dfetch(tex_val,j) *tex1Dfetch(tex_vec,col);
    }
    partialSums[t] = mySum;
    // Reduce partial sums
    if (id < 16) partialSums[t] += partialSums[t+16];
    if (id <  8) partialSums[t] += partialSums[t+ 8];
    if (id <  4) partialSums[t] += partialSums[t+ 4];
    if (id <  2) partialSums[t] += partialSums[t+ 2];
    if (id <  1) partialSums[t] += partialSums[t+ 1];

    // Write result 
    if (id == 0)
    {
      out[myRow] = partialSums[t];
    }
  }
}

int main(int argc, char **argv) {
  hipSetDevice(1);
  srand(2013);
  float *h_spmv_val, *h_spmv_valPad;
  int *h_spmv_cols, *h_spmv_colsPad;
  int *h_rowDelimiters, *h_rowDelimitersPad;
  float *h_spmv_vec, *h_spmv_out, *spmv_refOut;
  int spmv_nItems, nItemsPadded, spmv_numRows;

  spmv_numRows = MSIZE * (BLOCK_SIZE/WARP_SIZE);
  spmv_nItems = spmv_numRows * (spmv_numRows/ SFactor); 
  float maxval = 200.0;
  hipHostMalloc(&h_spmv_val, spmv_nItems * sizeof(float)); 
  hipHostMalloc(&h_spmv_cols, spmv_nItems * sizeof(int)); 
  hipHostMalloc(&h_rowDelimiters, (spmv_numRows + 1) * sizeof(int)); 
  fill(h_spmv_val, spmv_nItems, maxval); 
  initRandomMatrix(h_spmv_cols, h_rowDelimiters, spmv_nItems, spmv_numRows);

  // Set up remaining host data
  int paddedSize = spmv_numRows + (PAD_FACTOR - spmv_numRows % PAD_FACTOR);
  hipHostMalloc(&h_spmv_vec, spmv_numRows * sizeof(float)) ;
  spmv_refOut = new float[spmv_numRows];
  hipHostMalloc(&h_rowDelimitersPad, (spmv_numRows + 1) * sizeof(int)); 
  fill(h_spmv_vec, spmv_numRows, maxval);

  hipHostMalloc(&h_spmv_out, paddedSize * sizeof(float)); 
  convertToPadded(h_spmv_val, h_spmv_cols, spmv_numRows, h_rowDelimiters, &h_spmv_valPad,
          &h_spmv_colsPad, h_rowDelimitersPad, &nItemsPadded);

  // Compute reference solution
  spmvCpu(h_spmv_val, h_spmv_cols, h_rowDelimiters, h_spmv_vec, spmv_numRows, spmv_refOut);

  float *d_spmv_val, *d_spmv_vec, *d_spmv_out;
  int *d_spmv_cols, *d_rowDelimiters;

  // Allocate device memory
  hipMalloc(&d_spmv_val,  spmv_nItems * sizeof(float));
  hipMalloc(&d_spmv_cols, spmv_nItems * sizeof(int));
  hipMalloc(&d_spmv_vec,  spmv_numRows * sizeof(float));
  hipMalloc(&d_spmv_out,  spmv_numRows * sizeof(float));
  hipMalloc(&d_rowDelimiters, (spmv_numRows+1) * sizeof(int));

  // Transfer data to device
  hipMemcpy(d_spmv_val, h_spmv_val,   spmv_nItems * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_spmv_cols, h_spmv_cols, spmv_nItems * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_spmv_vec, h_spmv_vec, spmv_numRows * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rowDelimiters, h_rowDelimiters, (spmv_numRows+1) * sizeof(int), hipMemcpyHostToDevice);

  hipBindTexture(0,tex_vec,d_spmv_vec,spmv_numRows * sizeof(float));
  hipBindTexture(0,tex_val,d_spmv_val,spmv_nItems * sizeof(float));
  hipBindTexture(0,tex_row,d_rowDelimiters, (spmv_numRows+1) * sizeof(int));
  hipBindTexture(0,tex_col,d_spmv_cols,spmv_nItems * sizeof(int));  

  // Setup thread configuration
  int spmv_grid = (int) ceil(spmv_numRows / (float)(BLOCK_SIZE / WARP_SIZE));
// warm up the GPU
for(int i=0;i<5;i++)
{
  spmv_kernel <<<spmv_grid, BLOCK_SIZE>>>
  (d_spmv_val, d_spmv_cols, d_rowDelimiters, d_spmv_vec, spmv_numRows, d_spmv_out);
}

  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);

  // Setup thread configuration
for(int i=0;i<ITERATIONS;i++)
{
  spmv_kernel <<<spmv_grid, BLOCK_SIZE>>>
  (d_spmv_val, d_spmv_cols, d_rowDelimiters, d_spmv_vec, spmv_numRows, d_spmv_out);
}

  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time/ITERATIONS << endl;
  hipMemcpy(h_spmv_out, d_spmv_out, spmv_numRows * sizeof(float), hipMemcpyDeviceToHost);
//  spmv_verifyResults(spmv_refOut, h_spmv_out, spmv_numRows);

  // Don't forget to unbind texture memory
  hipUnbindTexture(tex_vec);
  hipUnbindTexture(tex_val);
  hipUnbindTexture(tex_row);
  hipUnbindTexture(tex_col);  

  return 0;
}

