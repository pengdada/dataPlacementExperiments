#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
//#include "../include/common.h"

#ifndef COMMON_H
#define COMMON_H

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <climits>


using namespace std;

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
        file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

inline bool
sdkCompareL2fe( const float* reference, const float* data,
                const unsigned int len, const float epsilon ) 
{
  assert( epsilon >= 0);

  float error = 0;
  float ref = 0;

  for( unsigned int i = 0; i < len; ++i) {

    float diff = reference[i] - data[i];
    error += diff * diff;
    ref += reference[i] * reference[i];
  }

  float normRef = sqrtf(ref);
  if (fabs(ref) < 1e-7) {
#ifdef _DEBUG
    std::cerr << "ERROR, reference l2-norm is 0\n";
#endif
    return false;
  }
  float normError = sqrtf(error);
  error = normError / normRef;
  bool result = error < epsilon;
#ifdef _DEBUG
  if( ! result) 
  {
    std::cerr << "ERROR, l2-norm error " 
      << error << " is greater than epsilon " << epsilon << "\n";
  }
#endif

  return result;
}

#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

__device__ uint get_smid(void) {
     uint ret;
     asm("mov.u32 %0, %smid;" : "=r"(ret) );
     return ret;
}

vector<int>::iterator randomMFromVec(vector<int>::iterator begin, vector<int>::iterator end, size_t num_random) {
  size_t left = std::distance(begin, end);
  while (num_random--) {
    vector<int>::iterator r = begin;
    std::advance(r, rand()%left);
    std::swap(*begin, *r);
    ++begin;
    --left;
  }
  return begin;
}

#endif

#define K 1
using namespace std;

//#define mm_BLOCK_SIZE_x 8
#define mm_BLOCK_SIZE 32
//#define mm_SUPER_BLOCKS_PER_SM 4
//int mm_SUPER_BLOCKS_PER_SM = 4;

#define iSizeMultiple 4 //must be multipes of 15

#define WA (12 * mm_BLOCK_SIZE) // Matrix A width
#define HA (12 * mm_BLOCK_SIZE) // Matrix A height
//#define WB (mm_SUPER_BLOCKS_PER_SM * mm_BLOCK_SIZE) // Matrix B width
#define WB (12 * mm_BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height

#define mm_GRID_X (WC*iSizeMultiple/mm_BLOCK_SIZE)
#define mm_GRID_Y (HC*iSizeMultiple/mm_BLOCK_SIZE)
#define mm_NBLOCKS (mm_GRID_X*mm_GRID_Y)

#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]

void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void
computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
  for (unsigned int i = 0; i < hA; ++i)
    for (unsigned int j = 0; j < wB; ++j) {
      double sum = 0;
      for (unsigned int k = 0; k < wA; ++k) {
        double a = A[i * wA + k];
        double b = B[k * wB + j];
        sum += a * b;
      }
      C[i * wB + j] = (float)sum;
    }
}
__global__ void
mm_kernel( float* C, float* A, float*   B, int wA, int wB)
{
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * mm_BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = mm_BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = mm_BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = mm_BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
      a <= aEnd;
      a += aStep, b += bStep) {

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[mm_BLOCK_SIZE][mm_BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[mm_BLOCK_SIZE][mm_BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    AS(ty, tx) = A[a + wA * ty + tx];
    BS(ty, tx) = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < mm_BLOCK_SIZE; ++k)
      Csub += AS(ty, k) * BS(k, tx);

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * mm_BLOCK_SIZE * by + mm_BLOCK_SIZE * bx;

  C[c + wB * ty + tx] = Csub;
//if (threadIdx.x==0&&threadIdx.y==0) atomicAdd(d_flag,1);

}

int main(int argc, char **argv) {
//  cudaSetDevice(1);
  srand(2013);
  unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;

  uiWA = WA * iSizeMultiple;
  uiHA = HA * iSizeMultiple;
  uiWB = WB * iSizeMultiple;
  uiHB = HB * iSizeMultiple;
  uiWC = WC * iSizeMultiple;
  uiHC = HC * iSizeMultiple;

  // allocate host memory for matrices A and B
  unsigned int size_A = uiWA * uiHA;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float* h_A = (float*)malloc(mem_size_A);
  unsigned int size_B = uiWB * uiHB;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float* h_B = (float*)malloc(mem_size_B);
//printf("size A = %d bytes,size B=%d bytes\n",mem_size_A,mem_size_B);
  // initialize host memory
  randomInit(h_A, size_A);
  randomInit(h_B, size_B);

  // allocate device memory
  float* d_A, *d_B, *d_C;
  unsigned int size_C = uiWC * uiHC;
  unsigned int mem_size_C = sizeof(float) * size_C;
printf("size A = %d bytes,size B=%d bytes,size C=%d bytes\n",mem_size_A,mem_size_B,mem_size_C);

  // allocate host memory for the result
  float* h_C      = (float*) malloc(mem_size_C);
  float* h_CUBLAS = (float*) malloc(mem_size_C);

  checkCudaErrors(hipMalloc((void**) &d_A, mem_size_A));
  checkCudaErrors(hipMalloc((void**) &d_B, mem_size_B));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
  checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );

  checkCudaErrors(hipMalloc((void**) &d_C, mem_size_C));

  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);
  // setup execution parameters
  dim3 mm_grid(mm_GRID_X, mm_GRID_Y);
  dim3 mm_block(mm_BLOCK_SIZE, mm_BLOCK_SIZE);
 // int mm_grid=mm_GRID_X*mm_GRID_Y;
  mm_kernel<<< mm_grid, mm_block>>>(d_C, d_A, d_B, uiWA, uiWB);


  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time << endl;
  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );

  // compute reference solution
  float* reference = (float*)malloc(mem_size_C);
  computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

  // check result (matrixMul)
  bool resCUDA = sdkCompareL2fe(reference, h_C, size_C, 1.0e-6f);
  printf("CUDA matrixMul compares %s\n\n", (true == resCUDA) ? "passed" : "FAIL");

//   ofstream f1("mm_correct.txt");
//   for(int i=0; i<size_C; ++i)
//     f1 << reference[i] << endl;
//   f1.close();
// 
//   ofstream f2("mm_gpu.txt");
//   for(int i=0; i<size_C; ++i)
//     f2 << h_C[i] << endl;
//   f2.close();


  // clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(reference);
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));


  return 0;
}

