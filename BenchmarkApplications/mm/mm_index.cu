#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#include "../include/common.h"
#define K 1
using namespace std;

#define mm_BLOCK_SIZE 16
//#define mm_SUPER_BLOCKS_PER_SM 4
//int mm_SUPER_BLOCKS_PER_SM = 4;

#define iSizeMultiple 4 //must be multipes of 15

#define WA (4 * mm_BLOCK_SIZE) // Matrix A width
#define HA (4 * mm_BLOCK_SIZE) // Matrix A height
//#define WB (mm_SUPER_BLOCKS_PER_SM * mm_BLOCK_SIZE) // Matrix B width
#define WB (60 * mm_BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height

#define mm_GRID_X (WC*iSizeMultiple/mm_BLOCK_SIZE)
#define mm_GRID_Y (HC*iSizeMultiple/mm_BLOCK_SIZE)
#define mm_NBLOCKS (mm_GRID_X*mm_GRID_Y)

#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]

void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void
computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
  for (unsigned int i = 0; i < hA; ++i)
    for (unsigned int j = 0; j < wB; ++j) {
      double sum = 0;
      for (unsigned int k = 0; k < wA; ++k) {
        double a = A[i * wA + k];
        double b = B[k * wB + j];
        sum += a * b;
      }
      C[i * wB + j] = (float)sum;
    }
}
__global__ void
mm_kernel( float* C, float* A, float* B, int wA, int wB, int *index_A,int *index_B)
{
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * mm_BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = mm_BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = mm_BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = mm_BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
      a <= aEnd;
      a += aStep, b += bStep) {

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[mm_BLOCK_SIZE][mm_BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[mm_BLOCK_SIZE][mm_BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    AS(ty, tx) = A[a + wA * ty + tx];
    BS(ty, tx) = B[b + wB * ty + tx];
if(blockIdx.x==0&&blockIdx.y==0&&threadIdx.x<4&&threadIdx.y<4)
{//printf("thread %d: access %d\n",threadIdx.y*4+threadIdx.x, a + wA * ty + tx);
 atomicAdd(&index_A[(threadIdx.y*4+threadIdx.x)*wA*wA+a+wA*ty+tx],1);
 atomicAdd(&index_B[(threadIdx.y*4+threadIdx.x)*wA*wB+b+wB*ty+tx],1);
}  
    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < mm_BLOCK_SIZE; ++k)
      Csub += AS(ty, k) * BS(k, tx);

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * mm_BLOCK_SIZE * by + mm_BLOCK_SIZE * bx;

  C[c + wB * ty + tx] = Csub;
//if (threadIdx.x==0&&threadIdx.y==0) atomicAdd(d_flag,1);

}

int main(int argc, char **argv) {
//  hipSetDevice(1);
  srand(2013);
  unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;

  uiWA = WA * iSizeMultiple;
  uiHA = HA * iSizeMultiple;
  uiWB = WB * iSizeMultiple;
  uiHB = HB * iSizeMultiple;
  uiWC = WC * iSizeMultiple;
  uiHC = HC * iSizeMultiple;

  // allocate host memory for matrices A and B
  unsigned int size_A = uiWA * uiHA;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float* h_A = (float*)malloc(mem_size_A);
  unsigned int size_B = uiWB * uiHB;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float* h_B = (float*)malloc(mem_size_B);

  // initialize host memory
  randomInit(h_A, size_A);
  randomInit(h_B, size_B);

  // allocate device memory
  float* d_A, *d_B, *d_C;
  unsigned int size_C = uiWC * uiHC;
  unsigned int mem_size_C = sizeof(float) * size_C;

  // allocate host memory for the result
  float* h_C      = (float*) malloc(mem_size_C);
  float* h_CUBLAS = (float*) malloc(mem_size_C);

  checkCudaErrors(hipMalloc((void**) &d_A, mem_size_A));
  checkCudaErrors(hipMalloc((void**) &d_B, mem_size_B));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
  checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );

  checkCudaErrors(hipMalloc((void**) &d_C, mem_size_C));

  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);
  // setup execution parameters
  dim3 mm_grid(mm_GRID_X, mm_GRID_Y);
  dim3 mm_block(mm_BLOCK_SIZE, mm_BLOCK_SIZE);
 // int mm_grid=mm_GRID_X*mm_GRID_Y;
  int *A_index=(int *)malloc(16*sizeof(int)*uiWA*uiHA);
  int *index_A,*index_B;
  int *B_index=(int *)malloc(16*sizeof(int)*uiWB*uiHB);
  hipMalloc((void **)&index_A,16*sizeof(int)*uiWA*uiHA);
  hipMalloc((void **)&index_B,16*sizeof(int)*uiWB*uiHB);
  hipMemset(index_A,0,16*sizeof(int)*uiHA*uiWA);
  hipMemset(index_B,0,16*sizeof(int)*uiHB*uiWB); 
 mm_kernel<<< mm_grid, mm_block>>>(d_C, d_A, d_B, uiWA, uiWB,index_A,index_B);


  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time << endl;
  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );
  hipMemcpy(A_index, index_A, 16*sizeof(int)*uiWA*uiWA, hipMemcpyDeviceToHost);
  hipMemcpy(B_index, index_B, 16*uiWA*uiWB*sizeof(int), hipMemcpyDeviceToHost);
   FILE *f1=fopen("A_D2.txt","w");
 for(int ii=0;ii<16;ii++){
    // fprintf(f2,"\n");
     for(int jj=0;jj<uiWA*uiWA;jj++)
    {if(A_index[ii*uiWA*uiWA+jj]!=0)
     fprintf(f1,"%d,%d ;",jj,A_index[ii*uiWA*uiWA+jj]);
   }
fprintf(f1,"\n");}
 FILE *f2=fopen("B_D2.txt","w");
 for(int ii=0;ii<16;ii++){
    // fprintf(f2,"\n");
     for(int jj=0;jj<uiWA*uiWB;jj++)
    {if(B_index[ii*uiWA*uiWB+jj]!=0)
     fprintf(f2,"%d,%d ;",jj,B_index[ii*uiWA*uiWB+jj]);
   }
fprintf(f2,"\n");}
  // compute reference solution
  float* reference = (float*)malloc(mem_size_C);
  computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

  // check result (matrixMul)
  bool resCUDA = sdkCompareL2fe(reference, h_C, size_C, 1.0e-6f);
  printf("CUDA matrixMul compares %s\n\n", (true == resCUDA) ? "passed" : "FAIL");

//   ofstream f1("mm_correct.txt");
//   for(int i=0; i<size_C; ++i)
//     f1 << reference[i] << endl;
//   f1.close();
// 
//   ofstream f2("mm_gpu.txt");
//   for(int i=0; i<size_C; ++i)
//     f2 << h_C[i] << endl;
//   f2.close();


  // clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(reference);
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
    printf("Profiling results saved to \"A_D2.txt\" and \"B_D2.txt\"\n");
    printf("Please use analysis.py to analysis them\n");

  return 0;
}

