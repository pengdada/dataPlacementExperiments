#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#include "../include/common.h"
#define K 1
using namespace std;


#define MSIZE 12*8*21 //22
#define BLOCK_SIZE 256
#define WARP_SIZE 32
surface<void,1> surf_vec;
static const double MAX_RELATIVE_ERROR = .02;

static const int PAD_FACTOR = 16;


void fill(float *A, const int n, const float maxi)
{
  for (int j = 0; j < n; j++) 
  {
    A[j] = ((float) maxi * (rand() / (RAND_MAX + 1.0f)));
  }
}

void initRandomMatrix(int *cols, int *rowDelimiters, const int n, const int dim)
{
  int nnzAssigned = 0;

  // Figure out the probability that a nonzero should be assigned to a given
  // spot in the matrix
  double prob = (double)n / ((double)dim * (double)dim);

  // Seed random number generator
  srand48(2013);

  // Randomly decide whether entry i,j gets a value, but ensure n values
  // are assigned
  bool fillRemaining = false;
  for (int i = 0; i < dim; i++)
  {
    rowDelimiters[i] = nnzAssigned;
    for (int j = 0; j < dim; j++)
    {
      int numEntriesLeft = (dim * dim) - ((i * dim) + j);
      int needToAssign   = n - nnzAssigned;
      if (numEntriesLeft <= needToAssign) {
        fillRemaining = true;
      }
      if ((nnzAssigned < n && drand48() <= prob) || fillRemaining)
      {
        // Assign (i,j) a value
        cols[nnzAssigned] = j;
        nnzAssigned++;
      }
    }
  }
  // Observe the convention to put the number of non zeroes at the end of the
  // row delimiters array
  rowDelimiters[dim] = n;
  assert(nnzAssigned == n);
}

void convertToPadded(float *A, int *cols, int dim, int *rowDelimiters, 
    float **newA_ptr, int **newcols_ptr, int *newIndices, 
    int *newSize) 
{
  // determine total padded size and new row indices
  int paddedSize = 0;  
  int rowSize; 

  for (int i=0; i<dim; i++) 
  {    
    newIndices[i] = paddedSize; 
    rowSize = rowDelimiters[i+1] - rowDelimiters[i]; 
    if (rowSize % PAD_FACTOR != 0) 
    {
      rowSize += PAD_FACTOR - rowSize % PAD_FACTOR; 
    } 
    paddedSize += rowSize; 
  }
  *newSize = paddedSize; 
  newIndices[dim] = paddedSize; 

  hipHostMalloc(newA_ptr, paddedSize * sizeof(float)); 
  hipHostMalloc(newcols_ptr, paddedSize * sizeof(int)); 

  float *newA = *newA_ptr; 
  int *newcols = *newcols_ptr; 

  memset(newA, 0, paddedSize * sizeof(float)); 

  // fill newA and newcols
  for (int i=0; i<dim; i++) 
  {
    for (int j=rowDelimiters[i], k=newIndices[i]; j<rowDelimiters[i+1]; 
        j++, k++) 
    {
      newA[k] = A[j]; 
      newcols[k] = cols[j]; 
    }
  }
}

void spmvCpu(const float *val, const int *cols, const int *rowDelimiters, 
    const float *vec, int dim, float *out) 
{
  for (int i=0; i<dim; i++) 
  {
    float t = 0; 
    for (int j = rowDelimiters[i]; j < rowDelimiters[i + 1]; j++)
    {
      int col = cols[j]; 
      t += val[j] * vec[col];
    }    
    out[i] = t; 
  }
}

void spmv_verifyResults(const float *cpuResults, const float *gpuResults,
    const int size) 
{
  bool passed = true; 
  for (int i = 0; i < size; i++)
  {
    if (fabs(cpuResults[i] - gpuResults[i]) / cpuResults[i] 
        > MAX_RELATIVE_ERROR) 
    {
      cout << "Failed! Mismatch at i: "<< i << " ref: " << cpuResults[i] <<
        " dev: " << gpuResults[i] << endl;
      return;
    }
  }

  cout << "spmv passed" << endl;
}
  __global__ void 
spmv_kernel(const float* val,
    const int    * cols,
    const int    * rowDelimiters,
    const float  * vec,
    const int dim, float * out)
{
  // Thread ID in block
  int t = threadIdx.x; 
  // Thread ID within warp
  int id = t & (WARP_SIZE-1);
  int warpsPerBlock = blockDim.x / WARP_SIZE;
  // One row per warp
  int myRow = (blockIdx.x * warpsPerBlock) + (t / WARP_SIZE);

  __shared__ volatile float partialSums[BLOCK_SIZE];

  if (myRow < dim) 
  {
    int warpStart = __ldg(&rowDelimiters[myRow]);
    int warpEnd = __ldg(&rowDelimiters[myRow+1]);
    float mySum = 0;
    for (int j = warpStart + id; j < warpEnd; j += WARP_SIZE)
    {
      float temp;
      int col = __ldg(&cols[j]); 
      surf1Dread(&temp,surf_vec,col*4,hipBoundaryModeTrap);      
      mySum += __ldg(&val[j]) * temp;//vec[col];
    }
    partialSums[t] = mySum;

    // Reduce partial sums
    if (id < 16) partialSums[t] += partialSums[t+16];
    if (id <  8) partialSums[t] += partialSums[t+ 8];
    if (id <  4) partialSums[t] += partialSums[t+ 4];
    if (id <  2) partialSums[t] += partialSums[t+ 2];
    if (id <  1) partialSums[t] += partialSums[t+ 1];

    // Write result 
    if (id == 0)
    {
      out[myRow] = partialSums[t];
    }
  }
}

int main(int argc, char **argv) {
  hipSetDevice(1);
  srand(2013);
  float *h_spmv_val, *h_spmv_valPad;
  int *h_spmv_cols, *h_spmv_colsPad;
  int *h_rowDelimiters, *h_rowDelimitersPad;
  float *h_spmv_vec, *h_spmv_out, *spmv_refOut;
  int spmv_nItems, nItemsPadded, spmv_numRows;

  spmv_numRows = MSIZE * (BLOCK_SIZE/WARP_SIZE);
  spmv_nItems = spmv_numRows * (spmv_numRows / 10); // 1% of entries will be non-zero
  float maxval = 200.0;
  hipHostMalloc(&h_spmv_val, spmv_nItems * sizeof(float)); 
  hipHostMalloc(&h_spmv_cols, spmv_nItems * sizeof(int)); 
  hipHostMalloc(&h_rowDelimiters, (spmv_numRows + 1) * sizeof(int)); 
  fill(h_spmv_val, spmv_nItems, maxval); 
  initRandomMatrix(h_spmv_cols, h_rowDelimiters, spmv_nItems, spmv_numRows);

  // Set up remaining host data
  int paddedSize = spmv_numRows + (PAD_FACTOR - spmv_numRows % PAD_FACTOR);
  hipHostMalloc(&h_spmv_vec, spmv_numRows * sizeof(float)) ;
  spmv_refOut = new float[spmv_numRows];
  hipHostMalloc(&h_rowDelimitersPad, (spmv_numRows + 1) * sizeof(int)); 
  fill(h_spmv_vec, spmv_numRows, maxval);

  hipHostMalloc(&h_spmv_out, paddedSize * sizeof(float)); 
  convertToPadded(h_spmv_val, h_spmv_cols, spmv_numRows, h_rowDelimiters, &h_spmv_valPad,
      &h_spmv_colsPad, h_rowDelimitersPad, &nItemsPadded);

  // Compute reference solution
  spmvCpu(h_spmv_val, h_spmv_cols, h_rowDelimiters, h_spmv_vec, spmv_numRows, spmv_refOut);

  float *d_spmv_val, *d_spmv_vec, *d_spmv_out;
  int *d_spmv_cols, *d_rowDelimiters;

  // Allocate device memory
  hipMalloc(&d_spmv_val,  spmv_nItems * sizeof(float));
  hipMalloc(&d_spmv_cols, spmv_nItems * sizeof(int));
  hipMalloc(&d_spmv_vec,  spmv_numRows * sizeof(float));
  hipMalloc(&d_spmv_out,  spmv_numRows * sizeof(float));
  hipMalloc(&d_rowDelimiters, (spmv_numRows+1) * sizeof(int));

  // Transfer data to device
  hipMemcpy(d_spmv_val, h_spmv_val,   spmv_nItems * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_spmv_cols, h_spmv_cols, spmv_nItems * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_spmv_vec, h_spmv_vec, spmv_numRows * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rowDelimiters, h_rowDelimiters, (spmv_numRows+1) * sizeof(int), hipMemcpyHostToDevice);
  //hipChannelFormatDesc channelDescA =  hipCreateChannelDesc<float>();
  hipChannelFormatDesc channelDescA=hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
  hipArray *A_vec;
  hipMallocArray(&A_vec,&channelDescA,spmv_numRows,1,hipArraySurfaceLoadStore);
  hipMemcpyToArray(A_vec,0,0,h_spmv_vec,spmv_numRows*sizeof(float),hipMemcpyHostToDevice);
  cudaBindSurfaceToArray(surf_vec,A_vec);
  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);

  // Setup thread configuration
  int spmv_grid = (int) ceil(spmv_numRows / (float)(BLOCK_SIZE / WARP_SIZE));
  for (int i=0; i<10; i++) // repeat 10 times
  spmv_kernel <<<spmv_grid, BLOCK_SIZE>>>
    (d_spmv_val, d_spmv_cols, d_rowDelimiters, d_spmv_vec, spmv_numRows, d_spmv_out);

  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds

  cout << "kernel exe time: " << kernel_time << endl;
  hipMemcpy(h_spmv_out, d_spmv_out, spmv_numRows * sizeof(float), hipMemcpyDeviceToHost);
  spmv_verifyResults(spmv_refOut, h_spmv_out, spmv_numRows);

  return 0;
}

